#include "hip/hip_runtime.h"
// slime/api/gpu_native.cu - Minimal API to launch the organism
#ifndef GPU_NATIVE_CU
#define GPU_NATIVE_CU
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../core/organism.cu"

// Forward declaration
__global__ void extract_ca_channel_kernel(float* output, float* concentration, int grid_size);

// Allocate and initialize organism on GPU
extern "C" Organism* create_organism() {
    Organism* h_organism = new Organism();
    Organism* d_organism;

    // Allocate organism structure on device
    hipMalloc(&d_organism, sizeof(Organism));

    // Allocate all components
    ComponentPool* d_pool;
    hipMalloc(&d_pool, sizeof(ComponentPool));
    hipMalloc(&d_pool->entries, MAX_POOL_SIZE * sizeof(PoolEntry));

    GPUElite* d_archive;
    hipMalloc(&d_archive, MAX_ARCHIVE_SIZE * sizeof(GPUElite));

    TemporalTube* d_tubes;
    hipMalloc(&d_tubes, sizeof(TemporalTube));
    hipMalloc(&d_tubes->entries, MAX_MEMORY_SIZE * sizeof(MemoryEntry));

    MultiHeadCAState* d_ca_state;
    hipMalloc(&d_ca_state, sizeof(MultiHeadCAState));
    hipMalloc(&d_ca_state->perception_weights, NUM_HEADS * CHANNELS * HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_ca_state->interaction_weights, NUM_HEADS * CHANNELS * HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_ca_state->value_weights, NUM_HEADS * HIDDEN_DIM * CHANNELS * sizeof(float));
    hipMalloc(&d_ca_state->head_mixing_weights, NUM_HEADS * NUM_HEADS * sizeof(float));
    hipMalloc(&d_ca_state->flow_kernels, NUM_HEADS * 9 * sizeof(float));
    hipMalloc(&d_ca_state->mass_buffer, NUM_HEADS * sizeof(float));

    BehavioralState* d_behavioral;
    hipMalloc(&d_behavioral, MAX_COMPONENTS * sizeof(BehavioralState));

    ChemicalField* d_chemical;
    hipMalloc(&d_chemical, sizeof(ChemicalField));
    hipMalloc(&d_chemical->concentration, GRID_SIZE * GRID_SIZE * sizeof(float));
    hipMalloc(&d_chemical->gradient_x, GRID_SIZE * GRID_SIZE * sizeof(float));
    hipMalloc(&d_chemical->gradient_y, GRID_SIZE * GRID_SIZE * sizeof(float));
    hipMalloc(&d_chemical->laplacian, GRID_SIZE * GRID_SIZE * sizeof(float));
    hipMalloc(&d_chemical->sources, GRID_SIZE * GRID_SIZE * sizeof(float));
    hipMalloc(&d_chemical->decay_factors, GRID_SIZE * GRID_SIZE * sizeof(float));

    // Allocate history buffers
    float* d_fitness_history;
    float* d_coherence_history;
    float* d_effective_rank_history;
    hipMalloc(&d_fitness_history, MAX_GENERATIONS * MAX_COMPONENTS * sizeof(float));
    hipMalloc(&d_coherence_history, MAX_GENERATIONS * MAX_COMPONENTS * sizeof(float));
    hipMalloc(&d_effective_rank_history, MAX_GENERATIONS * sizeof(float));

    // Set pointers in organism structure
    h_organism->pool = d_pool;
    h_organism->archive = d_archive;
    h_organism->memory_tubes = d_tubes;
    h_organism->ca_state = d_ca_state;
    h_organism->behavioral_agents = d_behavioral;
    h_organism->chemical_field = d_chemical;
    h_organism->fitness_history = d_fitness_history;
    h_organism->coherence_history = d_coherence_history;
    h_organism->effective_rank_history = d_effective_rank_history;

    // Copy organism structure to device
    hipMemcpy(d_organism, h_organism, sizeof(Organism), hipMemcpyHostToDevice);

    // Initialize with seed
    unsigned int seed = 42;
    init_organism_kernel<<<1, 1>>>(d_organism, seed);
    hipDeviceSynchronize();

    delete h_organism;
    return d_organism;
}

// Run organism for N generations
extern "C" void run_organism(Organism* d_organism, int generations) {
    printf("Starting organism evolution for %d generations...\n", generations);

    for (int gen = 0; gen < generations; gen++) {
        // Launch lifecycle with dynamic parallelism
        organism_lifecycle_kernel<<<1, 1>>>(d_organism, gen);
        hipDeviceSynchronize();

        // Print progress every 10 generations
        if (gen % 10 == 0) {
            // Get current stats
            Organism h_organism;
            hipMemcpy(&h_organism, d_organism, sizeof(Organism), hipMemcpyDeviceToHost);

            float fitness, coherence;
            hipMemcpy(&fitness, h_organism.fitness_history + gen * MAX_COMPONENTS,
                      sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(&coherence, h_organism.coherence_history + gen * MAX_COMPONENTS,
                      sizeof(float), hipMemcpyDeviceToHost);

            printf("Gen %4d: fitness=%.4f, coherence=%.4f\n", gen, fitness, coherence);

            // Check for convergence
            if (fitness > FITNESS_THRESHOLD && coherence > 0.9f) {
                printf("CONVERGED! Emergent behavior achieved.\n");
                break;
            }
        }
    }
}

// Cleanup organism
extern "C" void destroy_organism(Organism* d_organism) {
    // Get pointers from device
    Organism h_organism;
    hipMemcpy(&h_organism, d_organism, sizeof(Organism), hipMemcpyDeviceToHost);

    // Free all allocations
    hipFree(h_organism.pool->entries);
    hipFree(h_organism.pool);
    hipFree(h_organism.archive);
    hipFree(h_organism.memory_tubes->entries);
    hipFree(h_organism.memory_tubes);
    hipFree(h_organism.ca_state->perception_weights);
    hipFree(h_organism.ca_state->interaction_weights);
    hipFree(h_organism.ca_state->value_weights);
    hipFree(h_organism.ca_state->head_mixing_weights);
    hipFree(h_organism.ca_state->flow_kernels);
    hipFree(h_organism.ca_state->mass_buffer);
    hipFree(h_organism.ca_state);
    hipFree(h_organism.behavioral_agents);
    hipFree(h_organism.chemical_field->concentration);
    hipFree(h_organism.chemical_field->gradient_x);
    hipFree(h_organism.chemical_field->gradient_y);
    hipFree(h_organism.chemical_field->laplacian);
    hipFree(h_organism.chemical_field->sources);
    hipFree(h_organism.chemical_field->decay_factors);
    hipFree(h_organism.chemical_field);
    hipFree(h_organism.fitness_history);
    hipFree(h_organism.coherence_history);
    hipFree(h_organism.effective_rank_history);
    hipFree(d_organism);

    printf("Organism destroyed.\n");
}

// Get current CA state for visualization
extern "C" void get_ca_state(Organism* d_organism, float* h_buffer, int size) {
    Organism h_organism;
    hipMemcpy(&h_organism, d_organism, sizeof(Organism), hipMemcpyDeviceToHost);

    // Create temporary buffer for CA state
    float* d_ca_buffer;
    hipMalloc(&d_ca_buffer, size * sizeof(float));

    // Extract first channel of CA state
    dim3 grid((GRID_SIZE + 15) / 16, (GRID_SIZE + 15) / 16);
    dim3 block(16, 16);

    extract_ca_channel_kernel<<<grid, block>>>(
        d_ca_buffer,
        h_organism.chemical_field->concentration,
        GRID_SIZE
    );

    hipMemcpy(h_buffer, d_ca_buffer, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_ca_buffer);
}

// Helper kernel to extract CA channel
__global__ void extract_ca_channel_kernel(
    float* output,
    float* concentration,
    int grid_size
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < grid_size && y < grid_size) {
        output[y * grid_size + x] = concentration[y * grid_size + x];
    }
}

#endif // GPU_NATIVE_CU
