// Minimal test to verify CUDA compilation works
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void simple_kernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] *= 2.0f;
    }
}

int main() {
    printf("Testing minimal CUDA compilation...\n");

    // Allocate and test
    float* d_data;
    int size = 1024;
    hipMalloc(&d_data, size * sizeof(float));

    // Launch kernel
    simple_kernel<<<(size + 255) / 256, 256>>>(d_data, size);
    hipDeviceSynchronize();

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Success! CUDA is working.\n");

    hipFree(d_data);
    return 0;
}